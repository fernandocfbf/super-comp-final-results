#include "hip/hip_runtime.h"
// libraries
#include <random>
#include <iostream>
#include <vector> //to use vector
#include <string> //to use string
#include <algorithm> // max
#include<bits/stdc++.h>
#include <cmath>
#include <omp.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>

using namespace std;

#define MATCH 2
#define MISS -1
#define GAP  -1
#define BATCHSIZE 500000
#define MEM 500000

//globals
struct item {
    int score;
    vector<char> s1;
    vector<char> s2;
};
struct s_temp
{
    thrust::device_ptr<char> d_s;
    char letter_t;
    thrust::device_ptr<int> store_list_0;
    s_temp(
        thrust::device_ptr<char> d_s_,
        char letter_t_,
        thrust::device_ptr<int> store_list_0_) : 
        d_s(d_s_),
        letter_t(letter_t_),
        store_list_0(store_list_0_) {};
    __host__ __device__
    int operator() (const int(&j))
    {
        int w;
        if (d_s[j] == letter_t) {
            w = MATCH;
        }
        else {
            w = MISS;
        }

        int diagonal = store_list_0[j-1] + w;
        int insertion = store_list_0[j];
        int max = 0;
        if (diagonal > max) max = diagonal;
        if (insertion > max) max = insertion;
        return max;
    }
};

//functions sign
void generate_subsequence(vector<char> input, vector<vector<char>> &all);
int get_max_score(vector<vector<char>> &a_all, vector<vector<char>> &b_all);
int compute_score(vector<item> &batch_vector);
vector<item> slicer(vector<item> &arr, int X, int Y);

// functions code
vector<item> slicer(vector<item> &v, int m, int n){
    std::vector<item> vec(n - m + 1);
    std::copy(v.begin() + m, v.begin() + n + 1, vec.begin());
    return vec;
}
void generate_subsequence(string input, vector<vector<char>> &all){
    /*
    vector<char> sub_char(input.begin(), input.end());
    all.push_back(sub_char);
    */
    
    for (int j=0; j<(int)input.size(); j++){
        for(int i=j; i<(int)input.size(); i++){
            string sub = input.substr(i,j);
            vector<char> sub_char(sub.begin(), sub.end());
            if(sub_char.size() > 0){
                all.push_back(sub_char);
            }
        }
    }
    
}
int compute_score(vector<item> &batch_vector){
    int maximum_batch = -1;
    int i = 0;
    for (auto& pair_sub : batch_vector){
        i+=1;
        vector<char> S = pair_sub.s1;
        vector<char> T = pair_sub.s2;
        int N = (int)S.size();
        int M = (int)T.size();

        thrust::device_vector<int> store_list[2]; //store previous and current
        store_list[0].resize(N+1); //previous line
        store_list[1].resize(N+1); //current line

        thrust::fill(store_list[0].begin(), store_list[0].end(), 0);
        thrust::device_vector<char> d_S(N);
        thrust::copy(S.begin(), S.begin()+N, d_S.begin());

        thrust::counting_iterator<int> c0(1);
        thrust::counting_iterator<int> c1(M+1);

        for (int j=0; j<M; j++){
            char t_letter = T[j];

            //up and diagonal
            thrust::transform(c0, c1, store_list[1].begin()+1, s_temp(d_S.data(), t_letter, store_list[0].data()));
            
            //left side
            thrust::inclusive_scan(store_list[1].begin()+1, store_list[1].end(), store_list[0].begin()+1, thrust::maximum<int>());
        }
        int result_batch = (int)store_list[1].data()[store_list[1].size()-1];
        if (result_batch > maximum_batch){
            maximum_batch = result_batch;
        }
    }
    return maximum_batch;
}
int get_max_score(vector<vector<char>> &a_all, vector<vector<char>> &b_all){
    vector<item> subseq_pairs;
    int best_result_batch = -1;
    int best_result = -1;
    int score_computed = -1;
    int filled_spaces = 0;
    int computed = 0;

    for (auto& sub_a: a_all){        
        for (auto& sub_b: b_all){
            subseq_pairs.push_back({0, sub_a, sub_b});
            filled_spaces+=1;
            if (filled_spaces == MEM){
                #pragma omp parallel for reduction(max:best_result_batch) shared(computed)
                for (int i=0; i < subseq_pairs.size(); i+=BATCHSIZE){

                    vector<item> batch;

                    if (i + BATCHSIZE > (int)subseq_pairs.size()){
                        batch = slicer(subseq_pairs, i, (int)subseq_pairs.size()-1);
                    }
                    else{
                        batch = slicer(subseq_pairs, i, i+BATCHSIZE);
                    }
                    score_computed = compute_score(batch);
                    if (score_computed > best_result_batch){
                        best_result_batch = score_computed;
                    }
                    computed += BATCHSIZE;
                    //cout << "Computed " << i << "/" << subseq_pairs.size() << endl;
                    
                }
                if (best_result_batch > best_result){
                        best_result = best_result_batch;
                }
                filled_spaces = 0;
                subseq_pairs.clear();
            }
        }
    }
    #pragma omp parallel for reduction(max:best_result_batch)
    for (int i=0; i < subseq_pairs.size(); i+=BATCHSIZE){
        vector<item> batch;
        if (i + BATCHSIZE > (int)subseq_pairs.size()){
            batch = slicer(subseq_pairs, i, (int)subseq_pairs.size()-1);
        }
        else{
            batch = slicer(subseq_pairs, i, i+BATCHSIZE);
        }
        score_computed = compute_score(batch);
        if (score_computed > best_result_batch){
            best_result_batch = score_computed;
        }
        computed += BATCHSIZE;
        //cout << "Computed " << i << "/" << subseq_pairs.size() << endl;
    }
    if (best_result_batch > best_result){
            best_result = best_result_batch;
    }
    
    return best_result;
}

int main(){
    string S, T;
    int N, M;
    vector<vector<char>> get_all1, get_all2;

    cin >> N;
    cin >> M;
    cin >> S;
    cin >> T;

    generate_subsequence(S, get_all1);
    generate_subsequence(T, get_all2);

    int score_max = get_max_score(get_all1, get_all2);
    cout << score_max << endl;
    return 0;
}